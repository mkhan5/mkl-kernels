void func(void)
{
    double *d_A;
    hipMalloc(&d_A,      Nrows * Ncols * sizeof(double));
    hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice);

    // --- cuSOLVE input/output parameters/arrays
    int work_size = 0;
    int *devInfo;
    hipMalloc(&devInfo, sizeof(int));

    // --- CUDA solver initialization
    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    // --- CUDA CHOLESKY initialization
    hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_LOWER, Nrows, d_A, Nrows, &work_size);

    // --- CUDA POTRF execution
    double *work;
    hipMalloc(&work, work_size * sizeof(double));
    hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_LOWER, Nrows, d_A, Nrows, work, work_size, devInfo);
    int devInfo_h = 0;
   hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    if (devInfo_h != 0)
        printf("Unsuccessful potrf execution\n\n");

    // --- At this point, the upper triangular part of A contains the elements of L. Showing this.
    printf("\nFactorized matrix\n");
    hipMemcpy(h_A, d_A, Nrows * Ncols * sizeof(double), hipMemcpyDeviceToHost);

    hipsolverDnDestroy(solver_handle);
}

